#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <sys/time.h>

__global__ void recurse(int level)
{
    if (level<0)
    {
        int x = 1000;
        for(int i = 0; i<1e6; i++)
        {
            if (i%2)x*=10;
            else x/=10;
        }
        return;
    }
    level--;
    if (threadIdx.x == 0) recurse<<<1,2>>>(level);
    else recurse<<<1,2>>>(level);
    
    hipDeviceSynchronize(); //stop here in the current thread until the the above kernal is done

    __syncthreads(); //block level synchronization barrier from the parent kernal of this kernal
    return;
}

void serial_recurse(int level)
{
    if (level<0)
    {
        int x = 1000;
        for(int i = 0; i<1e6; i++)
        {
            if (i%2)x*=10;
            else x/=10;
        }
        return;
    }
    level--;
    serial_recurse(level);
    serial_recurse(level);
    return;

}


int main()
{
    struct timeval startwtime,endwtime;
    double seq_time;

    printf("Startin here\n");

    int level_1 = 10;
    gettimeofday(&startwtime,NULL);

    serial_recurse(level_1);

    gettimeofday(&endwtime,NULL);

    seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6+endwtime.tv_sec - startwtime.tv_sec);
    printf("serial time = %f \n", seq_time);

    int level=10;
    
    gettimeofday(&startwtime,NULL);

    recurse<<<1,2>>>(level);
    hipDeviceSynchronize();

    gettimeofday(&endwtime,NULL);

    seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6+endwtime.tv_sec - startwtime.tv_sec);
    printf("parralel time = %f \n", seq_time);

    return 0;
}