
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>// CUDA kernel. Each thread takes care of one element of c


using namespace std;

__global__ void fun(long long *a, long long n)
{
    // Get our global thread ID
    long long id = (long long)blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n and id%2==0)
        atomicAdd(reinterpret_cast<unsigned long long*>(a), 1);
}

int main(int argc, char *argv[])
{
    // Size of vectors
    long long n = 1e12;

    // Host input number
    long long *h_a;
    h_a = (long long *)malloc(sizeof(long long));
    *h_a = 0;

    // Device input number
    long long *d_a;

    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, sizeof(long long));

    // Copy host number to device
    hipMemcpy(d_a, h_a, sizeof(long long), hipMemcpyHostToDevice);

    long long blockSize, gridSize;

    // Number of threads in each thread block
    blockSize = 1024;

    // Number of thread blocks in grid
    gridSize = (long long)ceil((float)n / blockSize);

    cout << "Block Size: " << blockSize << ", Grid Size: " << gridSize << '\n';

    // Execute the kernel
    fun<<<gridSize, blockSize>>>(d_a, n);

    // Copy array back to host
    hipMemcpy(h_a, d_a, sizeof(long long), hipMemcpyDeviceToHost);

    ::cout << "Result: " << *h_a << '\n';

    // Release device memory
    hipFree(d_a);

    // Release host memory
    free(h_a);

    return 0;
}